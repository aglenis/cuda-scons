
#include <hip/hip_runtime.h>
//Minimal CUDA program
__global__ void foo(int* r) {
    if(threadIdx.x == 0) {
        r[0] = blockIdx.x;
    }
}


int main() {
    int* r;
    hipMalloc(&r, sizeof(int));
    foo<<<128, 128>>>(r);
}
